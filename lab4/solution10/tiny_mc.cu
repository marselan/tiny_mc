#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include "wtime.h"
#include "params.h"

using namespace std;

static void checkCudaCall(hipError_t statusCode) {
    if(statusCode != hipSuccess) {
        printf("Error: status code: %d\n", statusCode);
        exit(1);
    }
}

__device__ float next(uint64_t* randomNumber) {

    uint64_t a = 1103515245;
    uint64_t c = 12345;
    uint64_t m = (uint64_t)2<<30;
    float fm = (float)m;

    *randomNumber = *randomNumber * a + c;
    uint64_t s = *randomNumber >> 31;
    s = s << 31;
    *randomNumber = *randomNumber ^ s;
    return (float)*randomNumber / fm;
}

__global__ void photon(uint64_t* rnd, int threadCount) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i<threadCount) {

        uint64_t randomNumber = rnd[i];

        const float albedo = MU_S / (MU_S + MU_A);
        const float shells_per_mfp = 1e4 / MICRONS_PER_SHELL / (MU_A + MU_S);
        
        for(int k=0; k<PHOTONS_PER_THREAD; k++) {

            /* launch */
            float x = 0.0f;
            float y = 0.0f;
            float z = 0.0f;
            float u = 0.0f;
            float v = 0.0f;
            float w = 1.0f;
            float weight = 1.0f;

            for (;;) {
                float t = -logf(next(&randomNumber)); /* move */
                x += t * u;
                y += t * v;
                z += t * w;

                unsigned int shell = sqrtf(x * x + y * y + z * z) * shells_per_mfp; /* absorb */
                if (shell > SHELLS - 1) {
                    shell = SHELLS - 1;
                }
                //heat[shell] += (1.0f - albedo) * weight;
                //heat2[shell] += (1.0f - albedo) * (1.0f - albedo) * weight * weight; /* add up squares */
                weight *= albedo;

                /* New direction, rejection method */
                float xi1, xi2;
                do {
                    xi1 = 2.0f * next(&randomNumber) - 1.0f;
                    xi2 = 2.0f * next(&randomNumber) - 1.0f;
                    t = xi1 * xi1 + xi2 * xi2;
                } while (1.0f < t);
                u = 2.0f * t - 1.0f;
                v = xi1 * sqrtf((1.0f - u * u) / t);
                w = xi2 * sqrtf((1.0f - u * u) / t);

                if (weight < 0.001f) { /* roulette */
                    if (next(&randomNumber) > 0.1f) {
                        // exit
                        break;
                    }
                    weight /= 0.1f;
                }
            }
        }
        rnd[i] = randomNumber;
    }
}

int main() {

    double start = wtime();

    dim3 block(BLOCK_SIZE);
    dim3 grid((PHOTONS + block.x - 1) / block.x / PHOTONS_PER_THREAD);

    uint64_t* rnd = nullptr;

    int threadCount = PHOTONS / PHOTONS_PER_THREAD;
    checkCudaCall( hipMallocManaged(&rnd, threadCount * sizeof(uint64_t) ) );

    srand(SEED);
    for(int i=0; i<threadCount; i++) {
        rnd[i] = rand()>>2;
    }

    photon<<<grid, block>>>(rnd, threadCount);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipDeviceSynchronize());

    double end = wtime();
    double elapsed = end - start;
    
    cout<<PHOTONS<<"\t"<<elapsed<<"\t"<<(int)(1e-3 * PHOTONS / elapsed)<<endl;

    hipFree(rnd);

    return 0;
}